#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits>
#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>

#pragma hd_warning_disable
#define MAX_THREADS 16
#define FILENAME "data.txt"

typedef struct {
  int width;
  int height;
  int realWidth;
  int realHeight;
  int stride;
  float* elements;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

float GetElementCPU(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

void SetElementCPU(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}


void InitializeMatrices(Matrix& matA, int widthA, int heightA, int realWidthA, int realHeightA,
          Matrix& matB, int widthB, int heightB, int realWidthB, int realHeightB, std::istream& inputFile) {
  matA.width = widthA;
  matA.height = heightA;
  matA.realWidth = realWidthA;
  matA.realHeight = realHeightA;
  matA.stride = widthA;  // Assuming a simple row-major layout where stride equals width
  matA.elements = new float[widthA * heightA];

  matB.width = widthB;
  matB.height = heightB;
  matB.realWidth = realWidthB;
  matB.realHeight = realHeightB;
  matB.stride = widthB;  // Assuming a simple row-major layout where stride equals width
  matB.elements = new float[widthB * heightB];

  std::string inputString;
  std::string word;
  int i = 0;
  int j = 0;
  float value = 0.0f;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    j = 0;
    while (iss >> value) {
      SetElementCPU(matA, i, j, value);
      if(i < realWidthB){
        SetElementCPU(matB, j, i, value);
      }
      ++j;
    }
    ++i;
  }
  value = 0.0f;
  for (i = 0; i < heightA; ++i) {
    for (j = realWidthA; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = realHeightA; i < heightA; ++i) {
    for (j = 0; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = 0; i < heightB; ++i) {
    for (j = realWidthB; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
  for (i = realHeightB; i < heightB; ++i) {
    for (j = 0; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
}

void InitializeMatrix(Matrix& mat, int width, int height, int realWidth, int realHeight) {
  mat.width = width;
  mat.height = height;
  mat.realWidth = realWidth;
  mat.realHeight = realHeight;
  mat.stride = width;  // Assuming a simple row-major layout where stride equals width
  mat.elements = new float[width * height];
  float value = 0.0f;
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      SetElementCPU(mat, i, j, value);
    }
  }
}

int main() {

  std::ifstream inputFile(FILENAME);
  std::string inputString;
  getline(inputFile, inputString);
  int N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  int n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  int k = atoi(inputString.c_str()); // real B width, real C width

  int block_size;
    if(n <= k){ 
    block_size = min(MAX_THREADS, n);
    } else { // k < n
        block_size = min(MAX_THREADS, k);
    }

    int A_width = n;
    int B_height = n;
    int A_height = N;
    int B_width = k; 
    if(n % block_size != 0){
        A_width += (block_size - (n % block_size));
        B_height += (block_size - (n % block_size));
    }
    if(N % block_size != 0){
        A_height += (block_size - (N % block_size));
    }
    if(k % block_size != 0){
        B_width += (block_size - (k % block_size));
    }

  Matrix A, B, C;
  InitializeMatrices(A, A_width, A_height, n, N, B, B_width, B_height, k, n, inputFile);
  InitializeMatrix(C, B_width, A_height, k, N);
  inputFile.close();
  std::cout << "Matrix A:" << std::endl;
  for (int i = 0; i < A.height; ++i) {
    for (int j = 0; j < A.width; ++j) {
      std::cout << GetElementCPU(A, i, j) << " ";
    }
    std::cout << std::endl;
  }

  std::cout << "Matrix B:" << std::endl;
  for (int i = 0; i < B.height; ++i) {
    for (int j = 0; j < B.width; ++j) {
      std::cout << GetElementCPU(B, i, j) << " ";
    }
    std::cout << std::endl;
  }
  std::cout << "Matrix C:" << std::endl;
  for (int i = 0; i < C.height; ++i) {
    for (int j = 0; j < C.width; ++j) {
      std::cout << GetElementCPU(C, i, j) << " ";
    }
    std::cout << std::endl;
  }

  // Matrix d_A, d_B, d_C;
  // d_A.width = d_A.stride = A.width; 
  // d_A.height = A.height;
  // d_B.width = d_B.stride = B.width; 
  // d_B.height = B.height;
  // d_C.width = d_C.stride = C.width; 
  // d_C.height = C.height;
  // hipMalloc(&d_A.elements, A.width * A.height * sizeof(float));
  // hipMemcpy(d_A.elements, A.elements, A.width * A.height * sizeof(float),
  //            hipMemcpyHostToDevice);

  // hipMalloc(&d_B.elements, B.width * B.height * sizeof(float));
  // hipMemcpy(d_B.elements, B.elements, B.width * B.height * sizeof(float),
  //            hipMemcpyHostToDevice);

  // hipMalloc(&d_C.elements, C.width * C.height * sizeof(float));

  // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
  // dim3 dimGrid((int) ceil((double)B.width / (double)dimBlock.x),
  //              (int) ceil((double)A.height /(double) dimBlock.y)); 

  // unsigned long long time;
  // unsigned long long* d_time;
  // hipMalloc(&d_time, sizeof(unsigned long long));

  // MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_time); 
  // hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  // std::cout<<"Time: "<<time<<'\n';
  // hipMemcpy(C.elements, d_C.elements, C.width * C.height * sizeof(float),
  //            hipMemcpyDeviceToHost);
  // std::cout << "Matrix C:" << std::endl;
  // for (int i = 0; i < C.height; ++i) {
  //   for (int j = 0; j < C.width; ++j) {
  //     std::cout << GetElementCPU(C, i, j) << " ";
  //   }
  //   std::cout << std::endl;
  // }
  // hipFree(d_A.elements);
  // hipFree(d_B.elements);
  // hipFree(d_C.elements);
  // hipFree(d_time);
  return 0;
}