#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <limits>
#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>

#pragma hd_warning_disable
#define MAX_THREADS 16
#define FILENAME "data.txt"

typedef struct {
  int width;
  int height;
  int realWidth;
  int realHeight;
  int stride;
  float* elements;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

float GetElementCPU(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

void SetElementCPU(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}


void InitializeMatrices(Matrix& matA, int widthA, int heightA, int realWidthA, int realHeightA,
          Matrix& matB, int widthB, int heightB, int realWidthB, int realHeightB, std::istream& inputFile) {
  matA.width = widthA;
  matA.height = heightA;
  matA.realWidth = realWidthA;
  matA.realHeight = realHeightA;
  matA.stride = widthA;  // Assuming a simple row-major layout where stride equals width
  matA.elements = new float[widthA * heightA];

  matB.width = widthB;
  matB.height = heightB;
  matB.realWidth = realWidthB;
  matB.realHeight = realHeightB;
  matB.stride = widthB;  // Assuming a simple row-major layout where stride equals width
  matB.elements = new float[widthB * heightB];

  std::string inputString;
  std::string word;
  int i = 0;
  int j = 0;
  float value = 0.0f;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    j = 0;
    while (iss >> value) {
      SetElementCPU(matA, i, j, value);
      if(i < realWidthB){
        SetElementCPU(matB, j, i, value);
      }
      ++j;
    }
    ++i;
  }
  value = 0.0f;
  for (i = 0; i < heightA; ++i) {
    for (j = realWidthA; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = realHeightA; i < heightA; ++i) {
    for (j = 0; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = 0; i < heightB; ++i) {
    for (j = realWidthB; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
  for (i = realHeightB; i < heightB; ++i) {
    for (j = 0; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
}

void InitializeMatrix(Matrix& mat, int width, int height, int realWidth, int realHeight) {
  mat.width = width;
  mat.height = height;
  mat.realWidth = realWidth;
  mat.realHeight = realHeight;
  mat.stride = width;  // Assuming a simple row-major layout where stride equals width
  mat.elements = new float[width * height];
  float value = 0.0f;
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      SetElementCPU(mat, i, j, value);
    }
  }
}

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C, unsigned long long* time, int BLOCK_SIZE) {
    
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    float Cvalue = 0;
    int row = threadIdx.y;
    int col = threadIdx.x;

    unsigned long long startTime = clock();
   for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        __syncthreads();
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];
        __syncthreads();
    }
    SetElement(Csub, row, col, Cvalue);
    unsigned long long finishTime = clock();
    *time = (finishTime - startTime);
}

int main() {

  std::ifstream inputFile(FILENAME);
  std::string inputString;
  getline(inputFile, inputString);
  int N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  int n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  int k = atoi(inputString.c_str()); // real B width, real C width

  int block_size;
    if(n <= k){ 
    block_size = min(MAX_THREADS, n);
    } else { // k < n
        block_size = min(MAX_THREADS, k);
    }

    int A_width = n;
    int B_height = n;
    int A_height = N;
    int B_width = k; 
    if(n % block_size != 0){
        A_width += (block_size - (n % block_size));
        B_height += (block_size - (n % block_size));
    }
    if(N % block_size != 0){
        A_height += (block_size - (N % block_size));
    }
    if(k % block_size != 0){
        B_width += (block_size - (k % block_size));
    }

  Matrix A, B, C;
  InitializeMatrices(A, A_width, A_height, n, N, B, B_width, B_height, k, n, inputFile);
  InitializeMatrix(C, B_width, A_height, k, N);
  inputFile.close();
  std::cout << "Matrix A:" << std::endl;
  for (int i = 0; i < A.height; ++i) {
    for (int j = 0; j < A.width; ++j) {
      std::cout << GetElementCPU(A, i, j) << " ";
    }
    std::cout << std::endl;
  }

  std::cout << "Matrix B:" << std::endl;
  for (int i = 0; i < B.height; ++i) {
    for (int j = 0; j < B.width; ++j) {
      std::cout << GetElementCPU(B, i, j) << " ";
    }
    std::cout << std::endl;
  }
  std::cout << "Matrix C:" << std::endl;
  for (int i = 0; i < C.height; ++i) {
    for (int j = 0; j < C.width; ++j) {
      std::cout << GetElementCPU(C, i, j) << " ";
    }
    std::cout << std::endl;
  }

  Matrix d_A, d_B, d_C;
  d_A.width = d_A.stride = A.width; 
  d_A.height = A.height;
  d_A.realWidth = A.realWidth;
  d_A.realHeight = d_A.realHeight;
  d_B.width = d_B.stride = B.width; 
  d_B.height = B.height;
  d_B.realWidth = B.realWidth;
  d_B.realHeight = B.realHeight;
  d_C.width = d_C.stride = C.width; 
  d_C.height = C.height;
  d_C.realWidth = C.realWidth;
  d_C.realHeight = C.realHeight;
  hipMalloc(&d_A.elements, A.width * A.height * sizeof(float));
  hipMemcpy(d_A.elements, A.elements, A.width * A.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_B.elements, B.width * B.height * sizeof(float));
  hipMemcpy(d_B.elements, B.elements, B.width * B.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_C.elements, C.width * C.height * sizeof(float));

  dim3 dimBlock(block_size, block_size); 
  dim3 dimGrid((int) ceil((double)B.width / (double)dimBlock.x),
               (int) ceil((double)A.height /(double) dimBlock.y)); 

  unsigned long long time;
  unsigned long long* d_time;
  hipMalloc(&d_time, sizeof(unsigned long long));

  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_time, block_size); 
  hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  std::cout<<"Time: "<<time<<'\n';
  hipMemcpy(C.elements, d_C.elements, C.width * C.height * sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout << "Matrix C:" << std::endl;
  for (int i = 0; i < C.height; ++i) {
    for (int j = 0; j < C.width; ++j) {
      std::cout << GetElementCPU(C, i, j) << " ";
    }
    std::cout << std::endl;
  }
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
  hipFree(d_time);
  return 0;
}