#include <cfloat>
#include <climits>
#include <hip/hip_runtime.h>

#include <cmath>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <algorithm>

#pragma hd_warning_disable
//#define FILENAME "points_generated.txt"
#define FILENAME "data.txt"
#define BLOCK_SIZE 16
#define MAX_ITERATIONS 100
#define EPS 0.0001f
#define MAX_THREADS_IN_BLOCK 512

typedef struct {
  int width;
  int height;
  int realWidth;
  int realHeight;
  int stride;
  float* elements;
} Matrix;

__device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

float GetElementCPU(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

void SetElementCPU(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}


void InitializeMatrices(Matrix& matA, int widthA, int heightA, int realWidthA, int realHeightA,
          Matrix& matB, int widthB, int heightB, int realWidthB, int realHeightB, std::istream& inputFile) {
  matA.width = widthA;
  matA.height = heightA;
  matA.realWidth = realWidthA;
  matA.realHeight = realHeightA;
  matA.stride = widthA;  // Assuming a simple row-major layout where stride equals width
  matA.elements = new float[widthA * heightA];

  matB.width = widthB;
  matB.height = heightB;
  matB.realWidth = realWidthB;
  matB.realHeight = realHeightB;
  matB.stride = widthB;  // Assuming a simple row-major layout where stride equals width
  matB.elements = new float[widthB * heightB];

  std::string inputString;
  std::string word;
  int i = 0;
  int j = 0;
  float value = 0.0f;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    j = 0;
    while (iss >> value) {
      SetElementCPU(matA, i, j, value);
      if(i < realWidthB){
        SetElementCPU(matB, j, i, value);
      }
      ++j;
    }
    ++i;
  }
  value = 0.0f;
  for (i = 0; i < heightA; ++i) {
    for (j = realWidthA; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = realHeightA; i < heightA; ++i) {
    for (j = 0; j < widthA; ++j) {
      SetElementCPU(matA, i, j, value);
    }
  }
  for (i = 0; i < heightB; ++i) {
    for (j = realWidthB; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
  for (i = realHeightB; i < heightB; ++i) {
    for (j = 0; j < widthB; ++j) {
      SetElementCPU(matB, i, j, value);
    }
  }
}

void InitializeMatrix(Matrix& mat, int width, int height, int realWidth, int realHeight) {
  mat.width = width;
  mat.height = height;
  mat.realWidth = realWidth;
  mat.realHeight = realHeight;
  mat.stride = width;  // Assuming a simple row-major layout where stride equals width
  mat.elements = new float[width * height];
  float value = FLT_MAX;
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      SetElementCPU(mat, i, j, value);
    }
  }
}

 __device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

__global__ void KmeansKernel(Matrix A, Matrix B, Matrix C, unsigned long long* time) {
    
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
    float Cvalue = 0.0;
    int row = threadIdx.y;
    int col = threadIdx.x;

    unsigned long long startTime = clock();
   for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);
        __syncthreads();
        for (int e = 0; e < BLOCK_SIZE; ++e){
            Cvalue += pow(As[row][e] - Bs[e][col],2);
        }
        __syncthreads();
    }
    if(fabs(GetElement(Csub, row, col) - FLT_MAX) > EPS){
      SetElement(Csub, row, col, sqrt(Cvalue));
    }
    unsigned long long finishTime = clock();
    *time = (finishTime - startTime);
}

__global__ void MinInEachRow(Matrix C, int* result) {
  int rows = C.realHeight;
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // nr wiersza
      float minValue;
      int minIndex;
    if (tid < rows) {
      minValue = GetElement(C, tid, 0);
      minIndex = 0;
      for (int j = 0; j < C.realWidth; ++j) {
        if (GetElement(C, tid, j) < minValue) {
          minValue = GetElement(C, tid, j);
          minIndex = j;
        }
      } 
      result[tid] = minIndex;
    }
}

__global__ void CompareArrays(const int* array1, const int* array2, int size, int* count) {
    __shared__ int localCounts[MAX_THREADS_IN_BLOCK];

    int tid = threadIdx.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    localCounts[tid] = 0;
    for (int i = idx; i < size; i += gridSize) {
        if (array1[i] != array2[i]) {
            localCounts[tid]++;
        }
    }
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            localCounts[tid] += localCounts[tid + stride];
        }
        __syncthreads();
    }
    if (tid == 0) {
        atomicAdd(count, localCounts[0]);
    }
}

int main() {
  std::ifstream inputFile(FILENAME);
  std::string inputString;
  getline(inputFile, inputString);
  int N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  int n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  int k = atoi(inputString.c_str()); // real B width, real C width

    int A_width = n;
    int B_height = n;
    int A_height = N;
    int B_width = k; 
    if(n % BLOCK_SIZE != 0){
        A_width += (BLOCK_SIZE - (n % BLOCK_SIZE));
        B_height += (BLOCK_SIZE - (n % BLOCK_SIZE));
    }
    if(N % BLOCK_SIZE != 0){
        A_height += (BLOCK_SIZE - (N % BLOCK_SIZE));
    }
    if(k % BLOCK_SIZE != 0){
        B_width += (BLOCK_SIZE - (k % BLOCK_SIZE));
    }

  Matrix A, B, C;
  InitializeMatrices(A, A_width, A_height, n, N, B, B_width, B_height, k, n, inputFile);
  InitializeMatrix(C, B_width, A_height, k, N);
  inputFile.close();
  std::cout << "Matrix A:" << std::endl;
  for (int i = 0; i < A.realHeight; ++i) {
    for (int j = 0; j < A.realWidth; ++j) {
      std::cout << GetElementCPU(A, i, j) << " ";
    }
    std::cout << std::endl;
  }

  std::cout << "Matrix B:" << std::endl;
  for (int i = 0; i < B.realHeight; ++i) {
    for (int j = 0; j < B.realWidth; ++j) {
      std::cout << GetElementCPU(B, i, j) << " ";
    }
    std::cout << std::endl;
  }
  Matrix d_A, d_B, d_C;
  d_A.width = d_A.stride = A.width; 
  d_A.height = A.height;
  d_A.realWidth = A.realWidth;
  d_A.realHeight = d_A.realHeight;
  d_B.width = d_B.stride = B.width; 
  d_B.height = B.height;
  d_B.realWidth = B.realWidth;
  d_B.realHeight = B.realHeight;
  d_C.width = d_C.stride = C.width; 
  d_C.height = C.height;
  d_C.realWidth = C.realWidth;
  d_C.realHeight = C.realHeight;
  hipMalloc(&d_A.elements, A.width * A.height * sizeof(float));
  hipMemcpy(d_A.elements, A.elements, A.width * A.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_B.elements, B.width * B.height * sizeof(float));
  hipMemcpy(d_B.elements, B.elements, B.width * B.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_C.elements, C.width * C.height * sizeof(float));

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); 
  dim3 dimGrid((int) ceil((double)B.width / (double)dimBlock.x),
               (int) ceil((double)A.height /(double) dimBlock.y)); 

  unsigned long long time;
  unsigned long long* d_time;
  hipMalloc(&d_time, sizeof(unsigned long long));

  int* assignments = new int[N];
  std::fill(assignments, assignments + N, 0);
  int* d_assignments;
  hipMalloc(&d_assignments, N * sizeof(int));
  hipMemcpy(d_assignments, assignments, N * sizeof(int), hipMemcpyHostToDevice);

  int* newassignments = new int[N];
  std::fill(newassignments, newassignments + N, 0);
  int* d_newassignments;
  hipMalloc(&d_newassignments, N * sizeof(int));
  hipMemcpy(d_newassignments, newassignments, N * sizeof(int), hipMemcpyHostToDevice);

int numIters = 0;
int changes = INT_MAX;
int* d_changes;
hipMalloc(&d_changes, sizeof(int));
hipMemset(d_changes, 0, sizeof(int));

int gridSize = C.realHeight/MAX_THREADS_IN_BLOCK + 1;
std::cout<<"gridSize: "<<gridSize<<'\n';

while(numIters < 1 && (float)changes/(float)N > EPS){
  KmeansKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, d_time); 
  MinInEachRow<<<gridSize, MAX_THREADS_IN_BLOCK>>>(d_C, d_newassignments);
  //CompareArrays<<<gridSize, MAX_THREADS_IN_BLOCK>>>(d_newassignments, d_assignments, N, d_changes);
  hipMemcpy(newassignments, d_newassignments, N*sizeof(int), hipMemcpyDeviceToHost); // optional
  hipMemcpy(&changes, d_changes, sizeof(int), hipMemcpyDeviceToHost);
  ++numIters;
}
//std::cout << "\nNumber of different elements: " << changes << std::endl;
std::cout<< "Min in each row:\n";
for (int i=0; i<N; ++i) {
  std::cout<<newassignments[i]<<' ';
}
std:: cout<<'\n';

  hipMemcpy(&time, d_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  std::cout<<"Time: "<<time<<'\n';
  hipMemcpy(C.elements, d_C.elements, C.width * C.height * sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout << "Matrix C:" << std::endl;
  for (int i = 0; i < C.realHeight; ++i) {
    for (int j = 0; j < C.realWidth; ++j) {
      std::cout << GetElementCPU(C, i, j) << " ";
    }
    std::cout << std::endl;
  }
 

delete[] A.elements;
delete[] B.elements;
delete[] C.elements;
delete[] assignments;
delete[] newassignments;
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
  hipFree(d_time);
  hipFree(d_assignments);
  hipFree(d_changes);
  std::cout<<"\nBye!\n";
  return 0;
}