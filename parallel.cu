#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <climits>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdlib.h>
#include <string>


#pragma hd_warning_disable

#define BLOCK_SIZE 16
#define MAX_ITERATIONS 100
#define EPS 0.000001f
#define MAX_THREADS_IN_BLOCK 512
//#define MAX_THREADS_IN_BLOCK 16

typedef struct {
  int width;
  int height;
  int realWidth;
  int realHeight;
  int stride;
  float *elements;
} Matrix;

__host__ __device__ float GetElement(const Matrix A, int row, int col) {
  return A.elements[row * A.stride + col];
}

__host__ __device__ void SetElement(Matrix A, int row, int col, float value) {
  A.elements[row * A.stride + col] = value;
}

void FillMatrices(Matrix &matA, int widthA, int heightA, int realWidthA,
                        int realHeightA, Matrix &matB, int widthB, int heightB,
                        int realWidthB, int realHeightB,
                        std::istream &inputFile) {
  matA.width = widthA;
  matA.height = heightA;
  matA.realWidth = realWidthA;
  matA.realHeight = realHeightA;
  matA.stride = widthA; // Assuming a row-major layout, stride == width
  matA.elements = new float[widthA * heightA];

  matB.width = widthB;
  matB.height = heightB;
  matB.realWidth = realWidthB;
  matB.realHeight = realHeightB;
  matB.stride = widthB; // Assuming a row-major layout, stride == width
  matB.elements = new float[widthB * heightB];

  std::string inputString;
  int i = 0;
  int j = 0;
  float value = 0.0f;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    j = 0;
    while (iss >> value) {
      SetElement(matA, i, j, value);
      if (i < realWidthB) {
        SetElement(matB, j, i, value);
      }
      ++j;
    }
    ++i;
  }
  value = 0.0f;
  for (i = 0; i < heightA; ++i) {
    for (j = realWidthA; j < widthA; ++j) {
      SetElement(matA, i, j, value);
    }
  }
  for (i = realHeightA; i < heightA; ++i) {
    for (j = 0; j < widthA; ++j) {
      SetElement(matA, i, j, value);
    }
  }
  for (i = 0; i < heightB; ++i) {
    for (j = realWidthB; j < widthB; ++j) {
      SetElement(matB, i, j, value);
    }
  }
  for (i = realHeightB; i < heightB; ++i) {
    for (j = 0; j < widthB; ++j) {
      SetElement(matB, i, j, value);
    }
  }
}

void InitializeMatrix(Matrix &mat, int width, int height, int realWidth,
                      int realHeight) {
  mat.width = width;
  mat.height = height;
  mat.realWidth = realWidth;
  mat.realHeight = realHeight;
  mat.stride =
      width; // Assuming a row-major layout, stride == width
  mat.elements = new float[width * height];
  float value = FLT_MAX;
  for (int i = 0; i < height; ++i) {
    for (int j = 0; j < width; ++j) {
      SetElement(mat, i, j, value);
    }
  }
}

void InitializeDeviceMatrices(Matrix &A, Matrix &B, Matrix &C, Matrix &d_A, Matrix &d_B, Matrix &d_C){
  d_A.width = d_A.stride = A.width;
  d_A.height = A.height;
  d_A.realWidth = A.realWidth;
  d_A.realHeight = d_A.realHeight;
  d_B.width = d_B.stride = B.width;
  d_B.height = B.height;
  d_B.realWidth = B.realWidth;
  d_B.realHeight = B.realHeight;
  d_C.width = d_C.stride = C.width;
  d_C.height = C.height;
  d_C.realWidth = C.realWidth;
  d_C.realHeight = C.realHeight;
  hipMalloc(&d_A.elements, A.width * A.height * sizeof(float));
  hipMemcpy(d_A.elements, A.elements, A.width * A.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_B.elements, B.width * B.height * sizeof(float));
  hipMemcpy(d_B.elements, B.elements, B.width * B.height * sizeof(float),
             hipMemcpyHostToDevice);

  hipMalloc(&d_C.elements, C.width * C.height * sizeof(float));
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
  Matrix Asub;
  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.stride = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

__global__ void CalculateDistances(Matrix A, Matrix B, Matrix C) {

  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
  float Cvalue = 0.0;
  int row = threadIdx.y;
  int col = threadIdx.x;

  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
    Matrix Asub = GetSubMatrix(A, blockRow, m);
    Matrix Bsub = GetSubMatrix(B, m, blockCol);
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE]; // shared memory so every thread from one block reads from this 
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);
    __syncthreads(); // make sure the sub-matrices are loaded before starting the computation
    for (int e = 0; e < BLOCK_SIZE; ++e) {
      Cvalue += pow(As[row][e] - Bs[e][col], 2);
    }
    __syncthreads(); // make sure that computation is done
  }
  if (fabs(GetElement(Csub, row, col) - FLT_MAX) > EPS) {
    SetElement(Csub, row, col, sqrt(Cvalue));
  }
}

__global__ void MinInEachRow(Matrix C, int *result) {
  int rows = C.realHeight;
  int tid = threadIdx.x + blockIdx.x * blockDim.x; // number of row
  float minValue;
  int minIndex;
  if (tid < rows) {
    minValue = GetElement(C, tid, 0);
    minIndex = 0;
    for (int j = 0; j < C.realWidth; ++j) {
      if (GetElement(C, tid, j) < minValue) {
        minValue = GetElement(C, tid, j);
        minIndex = j;
      }
    }
    result[tid] = minIndex;
  }
}

__global__ void CompareArrays(const int *array1, const int *array2, int size,
                              int *count) {
  __shared__ int localCounts[MAX_THREADS_IN_BLOCK];

  int tid = threadIdx.x;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int gridSize = blockDim.x * gridDim.x;

  localCounts[tid] = 0;
  for (int i = idx; i < size; i += gridSize) {
    if (array1[i] != array2[i]) {
      localCounts[tid]++;
    }
  }
  __syncthreads(); // to wait for the shared memory operation to complete before continuing
  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      localCounts[tid] += localCounts[tid + stride];
    }
    __syncthreads(); // it is neccessary because in every iteratiion we use results from previous
  }
  if (tid == 0) { // to make sure that one block changes is added once
    atomicAdd(count, localCounts[0]); // to avoid parallelism problems
  }
}

__global__ void ComputeAverage(Matrix B, const int *numOfVectors, int k,
                               int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < k) {
    for (int i = 0; i < n; ++i) {
      SetElement(B, i, tid,
                 (float)GetElement(B, i, tid) / (float)numOfVectors[tid]);
    }
  }
}

__global__ void ComputeSum(Matrix matA, const int *groups, Matrix matB, int N,
                           int k, int n, int *numOfVectors) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    int groupId = groups[tid];
    atomicAdd(&numOfVectors[groupId], 1);
    __syncthreads(); // make sure that every thread incremented numOfVectors[groupId]
    for (int i = 0; i < n; ++i) {
      atomicAdd(&matB.elements[i * matB.stride + groupId],
                GetElement(matA, tid, i));
    }
  }
  //__syncthreads();
}

void readFile(std::istream &inputFile, int& N, int& n, int& k, Matrix& A, Matrix& B, Matrix& C){
  std::string inputString;
  getline(inputFile, inputString);
  N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  k = atoi(inputString.c_str()); // real B width, real C width

  // A is N*n, but I want be able to split A into full blocks, so I want the height and the width be divisible by BLOCK_SIZE
  // Same for B (n*k) and C (N*k)
  int A_width = n;
  int B_height = n;
  int A_height = N;
  int B_width = k;
  if (n % BLOCK_SIZE != 0) {
    A_width += (BLOCK_SIZE - (n % BLOCK_SIZE));
    B_height += (BLOCK_SIZE - (n % BLOCK_SIZE));
  }
  if (N % BLOCK_SIZE != 0) {
    A_height += (BLOCK_SIZE - (N % BLOCK_SIZE));
  }
  if (k % BLOCK_SIZE != 0) {
    B_width += (BLOCK_SIZE - (k % BLOCK_SIZE));
  }


  // Read data into matrices
  FillMatrices(A, A_width, A_height, n, N, B, B_width, B_height, k, n,
                     inputFile);
  // Matrix A contains dataset: one row - one vektor
  // Matrix B contains k centroids (first k vectors from dataset): one column - one vector 
  InitializeMatrix(C, B_width, A_height, k, N); // C will contain distances
}

void defineArrays(int& N, int& k, int*& assignments, int*& d_assignments, int*& newassignments, 
                  int*& d_newassignments, int*& numOfVectorsInClusters, int*& d_numOfVectorsInClusters, int*& d_changes){ 
  assignments = new int[N];
  std::fill(assignments, assignments + N, 0);
  hipMalloc(&d_assignments, N * sizeof(int));
  hipMemcpy(d_assignments, assignments, N * sizeof(int),
             hipMemcpyHostToDevice);

  newassignments = new int[N];
  std::fill(newassignments, newassignments + N, 0);
  hipMalloc(&d_newassignments, N * sizeof(int));
  hipMemcpy(d_newassignments, newassignments, N * sizeof(int),
             hipMemcpyHostToDevice);

  numOfVectorsInClusters = new int[k];
  std::fill(numOfVectorsInClusters, numOfVectorsInClusters + k, 0);
  hipMalloc(&d_numOfVectorsInClusters, k * sizeof(int));
  hipMemset(d_numOfVectorsInClusters, 0, k * sizeof(int));
  
  hipMalloc(&d_changes, sizeof(int));
  hipMemset(d_changes, 0, sizeof(int));
}

void freeMemory(Matrix& A, Matrix& B, Matrix& C,
               int* assignments,int* newassignments,int* numOfVectorsInClusters, 
               Matrix& d_A, Matrix& d_B, Matrix& d_C,
               int* d_assignments,int* d_newassignments,int* d_numOfVectorsInClusters,int* d_changes){
  delete[] A.elements;
  delete[] B.elements;
  delete[] C.elements;
  delete[] assignments;
  delete[] newassignments;
  delete[] numOfVectorsInClusters;
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
  hipFree(d_assignments);
  hipFree(d_newassignments);
  hipFree(d_numOfVectorsInClusters);
  hipFree(d_changes);
}


KMeansClusterization(int& N, int& n, int& k, Matrix& A, Matrix& B, Matrix& C, Matrix& d_A, Matrix& d_B, Matrix& d_C){
  hipEvent_t  startStage, stopStage;
  int numIters = 0; // number of iterations
  int changes = INT_MAX; // number of vectors that changed cluster during last iteration
  int *assignments, *d_assignments, *newassignments, *d_newassignments,  *numOfVectorsInClusters, *d_numOfVectorsInClusters, *d_changes;
  float elapsedTimeFullAlgoritm, tmpTime, elapsedTimeCalcDist = 0.0, elapsedTimeFindMin = 0.0, elapsedTimeComapreArrays = 0.0, elapsedTimeComputeAverage = 0.0;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((int)ceil((double)B.width / (double)dimBlock.x),
               (int)ceil((double)A.height / (double)dimBlock.y));

  // assignments (size N): assignments[i] == old cluster number for i-th vector 
  // newassignments (size N): newassignments[i] == new cluster number for i-th vector 
  // numOfVectorsInClusters (size k): numOfVectorsInClusters[i] == number of vectors in i-th cluster 
  defineArrays(N, k, assignments, d_assignments, newassignments, d_newassignments, numOfVectorsInClusters, d_numOfVectorsInClusters, d_changes);

  int gridSize = C.realHeight / MAX_THREADS_IN_BLOCK + 1;
  
  hipEventCreate(&startStage);
  hipEventCreate(&stopStage);
  while (numIters < MAX_ITERATIONS && (float)changes / (float)N > EPS) {
    hipEventRecord(startStage,0);
    CalculateDistances<<<dimGrid, dimBlock>>>(d_A, d_B, d_C); // C[i,j] - distance between i-th vector and j-th centroid
    hipEventRecord(stopStage,0);
    hipEventSynchronize(stopStage);
    hipEventElapsedTime(&tmpTime,startStage,stopStage);
    elapsedTimeCalcDist += tmpTime;

    hipMemset(d_B.elements, 0.0, d_B.height * d_B.width * sizeof(float));
    hipMemset(d_numOfVectorsInClusters, 0, k * sizeof(int));
    hipMemset(d_changes, 0, sizeof(int));

    
    hipEventRecord(startStage,0);
    MinInEachRow<<<gridSize, MAX_THREADS_IN_BLOCK>>>(d_C, d_newassignments);
    hipEventRecord(stopStage,0);
    hipEventSynchronize(stopStage);
    hipEventElapsedTime(&tmpTime,startStage,stopStage);
    elapsedTimeFindMin += tmpTime;

    hipEventRecord(startStage,0);
    CompareArrays<<<gridSize, MAX_THREADS_IN_BLOCK>>>(
        d_newassignments, d_assignments, N, d_changes);
    hipEventRecord(stopStage,0);
    hipEventSynchronize(stopStage);
    hipEventElapsedTime(&tmpTime,startStage,stopStage);
    elapsedTimeComapreArrays += tmpTime;

    hipEventRecord(startStage,0);
    ComputeSum<<<gridSize, MAX_THREADS_IN_BLOCK>>>(
        d_A, d_newassignments, d_B, N, k, n, d_numOfVectorsInClusters);

    ComputeAverage<<<gridSize, MAX_THREADS_IN_BLOCK>>>(
        d_B, d_numOfVectorsInClusters, k, n);
    hipEventRecord(stopStage,0);
    hipEventSynchronize(stopStage);
    hipEventElapsedTime(&tmpTime,startStage,stopStage);
    elapsedTimeComputeAverage += tmpTime;

    hipMemcpy(d_assignments, d_newassignments, N * sizeof(int),
               hipMemcpyDeviceToDevice);
    hipMemcpy(&changes, d_changes, sizeof(int), hipMemcpyDeviceToHost);
    ++numIters;
  }
  std::cout<<"Elapsed Time [Distance calculation stage] = "<<elapsedTimeCalcDist<<" milliseconds\n";
  std::cout<<"Elapsed Time [Finding minimum stage] = "<<elapsedTimeFindMin<<" milliseconds\n";
  std::cout<<"Elapsed Time [Array comparing stage] = "<<elapsedTimeComapreArrays<<" milliseconds\n";
  std::cout<<"Elapsed Time [Computing average stage] = "<<elapsedTimeComputeAverage<<" milliseconds\n";   
  
  hipEventDestroy(startStage);
  hipEventDestroy(stopStage);               
}

int main(int argc, char** argv) {
  // file validation
  std::string inFile = "";
    if( argc == 2 ) {
      inFile = argv[1];
    }
    else {
      std::cout << "Usage: ./cufile InputFile \n";
      return 1;
    }
  std::ifstream inputFile;
  inputFile.open(inFile.c_str(), std::ios::in);
  if (!inputFile.is_open()) {
        std::cout << "Error opening file: " << inFile << std::endl;
        return 1;
    }

  // data declaration
  Matrix A, B, C, d_A, d_B, d_C; 
  int N, n, k;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  // data initialization from file
  hipEventRecord(start,0);
  readFile(inputFile, N, n, k, A, B, C);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  std::cout<<"\nElapsed Time [Data reading] = "<<elapsedTime<<" milliseconds\n";
  inputFile.close();

  hipEventRecord(&start,0);
  InitializeDeviceMatrices(A, B, C, d_A, d_B, d_C);
  hipEventRecord(&stop,0);
  hipEventSynchronize(&stop);
  hipEventElapsedTime(&elapsedTime, &start, &stop);
  std::cout<<"Elapsed Time [CPU - GPU copying] = "<<elapsedTime<<" milliseconds\n";

  // K-means clusterization
  hipEventRecord(&start,0);
  KMeansClusterization(N, n, k, A, B, C, d_A, d_B, d_C);
  hipEventRecord(&stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime,start,stop);
  std::cout<<"Elapsed Time [Full algorithm + time measurement] = "<<elapsedTimeFullAlgoritm<<" milliseconds\n";



  hipMemcpy(B.elements, d_B.elements, B.width * B.height * sizeof(float),
             hipMemcpyDeviceToHost);
  std::cout<<"Iterations:"<< numIters<<'\n';
  std::cout << "Centroids:" << std::endl;
  for (int i = 0; i < B.realHeight; ++i) {
    for (int j = 0; j < B.realWidth; ++j) {
      std::cout << GetElement(B, i, j) << " ";
    }
    std::cout << std::endl;
  }
  
  freeMemory(A, B, C, assignments, newassignments, numOfVectorsInClusters, d_A, d_B, d_C, d_assignments, d_newassignments, d_numOfVectorsInClusters, d_changes);
  hipEventDestroy(start); 
  hipEventDestroy(stop);
  return 0;
}