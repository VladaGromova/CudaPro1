#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <limits.h>
#include <sstream>
#include <stdlib.h>
#include <string>
//#include <sys/time.h>
#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#pragma hd_warning_disable

#define MAX_ITERATIONS 100
#define EPS 0.000001f


struct dkeygen : public thrust::unary_function<int, int> {
  int dim;
  int numd;

  dkeygen(const int _dim, const int _numd) : dim(_dim), numd(_numd){};

  __host__ __device__ int operator()(const int val) const {
    return (val / dim);
  }
};

struct clusterkeygen : public thrust::unary_function<int, int> {
  int stride;

  clusterkeygen(const int _stride) : stride(_stride){};

  __host__ __device__ int operator()(const int val) const {
    return (val % stride);
  }
};

typedef thrust::tuple<float, float> mytuple;
struct my_dist : public thrust::unary_function<mytuple, float> {
  __host__ __device__ float operator()(const mytuple &my_tuple) const {
    float temp = thrust::get<0>(my_tuple) - thrust::get<1>(my_tuple);
    return temp * temp;
  }
};

struct MinWithIndex {
  __host__ __device__ thrust::tuple<float, int>
  operator()(const thrust::tuple<float, int> &a,
             const thrust::tuple<float, int> &b) const {
    return (thrust::get<0>(a) < thrust::get<0>(b)) ? a : b;
  }
};

struct d_idx : public thrust::unary_function<int, int> {
  int dim;
  int numd;

  d_idx(int _dim, int _numd) : dim(_dim), numd(_numd){};

  __host__ __device__ int operator()(const int val) const {
    return (val % (dim * numd));
  }
};

struct c_idx : public thrust::unary_function<int, int> {
  int dim;
  int numd;

  c_idx(int _dim, int _numd) : dim(_dim), numd(_numd){};

  __host__ __device__ int operator()(const int val) const {
    return (val % dim) + (dim * (val / (dim * numd)));
  }
};

struct my_sqrt : public thrust::unary_function<float, float> {
  __host__ __device__ float operator()(const float val) const {
    return sqrtf(val);
  }
};

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T, T> {
  T C; // number of columns

  __host__ __device__ linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__ T operator()(T i) { return i / C; }
};

struct sum_functor {
  __host__ __device__ float operator()(const float &a, const float &b) const {
    return a + b;
  }
};

struct div_functor : public thrust::unary_function<int, int> {
  int m;
  div_functor(int _m) : m(_m){};

  __host__ __device__ int operator()(int x) const { return x / m; }
};

struct is_true {
  __host__ __device__ bool operator()(bool x) { return x; }
};

struct centr_sum_functor {
  int R;
  int C;
  float *arr;

  centr_sum_functor(int _R, int _C, float *_arr) : R(_R), C(_C), arr(_arr){};

  __host__ __device__ float operator()(int myC) {
    float sum = 0.0;
    for (int i = 0; i < R; i++)
      sum += arr[i * C + myC];
    return sum;
  }
};

struct NotEqual {
  __host__ __device__ int operator()(thrust::tuple<int, int> t) const {
    return thrust::get<0>(t) != thrust::get<1>(t) ? 1 : 0;
  }
};

void calculateDistances(int &n, int &N, int &k,
                        thrust::device_vector<float> &d_data,
                        thrust::device_vector<float> &d_centr,
                        thrust::device_vector<float> &tmp_distances) {
  // we want to imitate this structure: (c - centroids, v - vetors)
  // c1 c2 ... ck | c1 c2 ... ck | ...
  // v1 v1 ... v1 | v2 v2 ... v2 | ...
  // => d(v1,c1) ... d(v1,cn) | ....
  // => min_dist_v1_cluster | min_dist_v2_cluster | ...
  thrust::reduce_by_key(
      // keys: 0...0 1...1 ... k*N
      thrust::make_transform_iterator(
          thrust::make_counting_iterator<int>(0),
          dkeygen(n, N)), // mod n (begining of input key range)
      thrust::make_transform_iterator(
          thrust::make_counting_iterator<int>(n * N * k),
          dkeygen(n, N)), // (end of input key range)
      thrust::make_transform_iterator(
          thrust::make_zip_iterator( // 
              thrust::make_tuple(
                  thrust::make_permutation_iterator(
                      d_centr.begin(),
                      thrust::make_transform_iterator(
                          thrust::make_counting_iterator<int>(0), d_idx(n, k))), // function to take coordinate of centroid
                  thrust::make_permutation_iterator(
                      d_data.begin(),
                      thrust::make_transform_iterator(
                          thrust::make_counting_iterator<int>(0), 
                          c_idx(n, k))))), // to take coordinate of vector 
          my_dist()),
      thrust::make_discard_iterator(), // keys output, we don't need it
      tmp_distances.begin()               // values output - result (distances)
  );

  thrust::transform(tmp_distances.begin(), tmp_distances.end(), tmp_distances.begin(),
                    my_sqrt());
}

void findNearestCentroid(int &k, int &N, thrust::device_vector<float> &d_centr,
                         thrust::device_vector<float> &tmp_distances,
                         thrust::device_vector<float> &mins,
                         thrust::device_vector<float> &vec_modulus_k,
                         thrust::device_vector<int> &d_clusters) {
  // if we interprate it lika a matrix the task will be just to find minimum for each row 
  thrust::reduce_by_key(
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      linear_index_to_row_index<int>(k)), // to get the number of element in row 
                                      //-> it gives us cluster number (but from the begining of array, not row)
      thrust::make_transform_iterator(thrust::counting_iterator<int>(k * N),
                                      linear_index_to_row_index<int>(k)),
      thrust::make_zip_iterator(thrust::make_tuple(
          tmp_distances.begin(), thrust::counting_iterator<int>(0))),
      thrust::make_discard_iterator(), // we don;'t need keys output
      thrust::make_zip_iterator(
          thrust::make_tuple(mins.begin(), d_clusters.begin())),
      thrust::equal_to<int>(), MinWithIndex());

  thrust::fill(vec_modulus_k.begin(), vec_modulus_k.end(), k);
  thrust::transform(d_clusters.begin(), d_clusters.end(), vec_modulus_k.begin(),
                    d_clusters.begin(), thrust::modulus<int>()); // to get real number of centroid 
}

void countClusterChanges(int &delta, thrust::device_vector<int> &old_d_clusters,
                         thrust::device_vector<int> &d_clusters) { // difference between previous assignment and actual
  delta = thrust::transform_reduce(
      thrust::make_zip_iterator(
          thrust::make_tuple(old_d_clusters.begin(), d_clusters.begin())),
      thrust::make_zip_iterator(
          thrust::make_tuple(old_d_clusters.end(), d_clusters.end())),
      NotEqual(), 0, thrust::plus<int>());
}

void findNewCentroids(int &n, int &N, int &k,
                      thrust::device_vector<float> &d_data,
                      thrust::device_vector<float> &d_centr,
                      thrust::device_vector<int> &indices,
                      thrust::device_vector<int> &d_clusters,
                      thrust::device_vector<int> &clusterSizes,
                      thrust::device_vector<int> &data_starts,
                      thrust::device_vector<int> &data_ends,
                      thrust::device_vector<float> &vectorsInCluster,
                      thrust::device_vector<float> &actual_indices,
                      thrust::device_vector<float> &fcol_sums,
                      thrust::device_vector<bool> &docopy) {
  thrust::sequence(indices.begin(), indices.end());
  // if in 1st cluster there are v0, v5, in 2nd - v1, ... then indices will be [0, 5, 1, ...]
  thrust::sort_by_key(d_clusters.begin(), d_clusters.end(), indices.begin());

  // num of vectors in each cluster
  thrust::reduce_by_key(d_clusters.begin(), d_clusters.end(),
                        thrust::make_constant_iterator(1), // we have to add 1 to sum if vector is in cluster
                        thrust::make_discard_iterator(), clusterSizes.begin(),
                        thrust::equal_to<int>(), thrust::plus<int>());
  
  thrust::fill(d_centr.begin(), d_centr.end(), 0.0);
  // for each cluster we want to find range of indices (vector numbers-ids) - actual_indices
  thrust::exclusive_scan(clusterSizes.begin(), clusterSizes.end(),
                         data_starts.begin()); 
  thrust::inclusive_scan(clusterSizes.begin(), clusterSizes.end(),
                         data_ends.begin());
  
  for (int i = 0; i < k; ++i) {
    vectorsInCluster.resize(clusterSizes[i] * n); // vectors in i-th cluster
    actual_indices.resize(clusterSizes[i]);
    thrust::copy(indices.begin() + data_starts[i], indices.end() + data_ends[i],
                 actual_indices.begin()); // cut actual segment
    
    thrust::binary_search(
        actual_indices.begin(), actual_indices.end(),
        thrust::make_transform_iterator(thrust::make_counting_iterator(0),
                                        div_functor(n)), // mod n
        thrust::make_transform_iterator(thrust::make_counting_iterator(0),
                                        div_functor(n)) +
            N * n,
        docopy.begin());
  
    thrust::copy_if(d_data.begin(), d_data.end(), docopy.begin(), // get actual vectors
                    vectorsInCluster.begin(), is_true());
    
    thrust::sequence(fcol_sums.begin(), fcol_sums.end());
    
    thrust::transform(
        fcol_sums.begin(), fcol_sums.end(), d_centr.begin() + i * n,
        centr_sum_functor(clusterSizes[i], n, // sum up elements from same columns
                          thrust::raw_pointer_cast(vectorsInCluster.data())));
    
    hipDeviceSynchronize(); // reason: raw_pointer
    thrust::transform(d_centr.begin() + i * n, d_centr.begin() + (i + 1) * n,
                      thrust::make_constant_iterator(clusterSizes[i]),
                      d_centr.begin() + i * n, thrust::divides<float>());
    
  }
  
}


void KMeansClustering(float *&data, float *&cs, int *&clstrs, int k, int n,
                      int N, int print) {

  // additional data declaration
  thrust::device_vector<float> tmp_distances(k * N); // distances for k centroids and N vectors
  int delta = INT_MAX;
  int numIters = 0;
  thrust::device_vector<int> d_clusters(N); // vector for cluster assignments
  thrust::device_vector<int> old_d_clusters(N); // vector for previous cluster assignments
  thrust::fill(d_clusters.begin(), d_clusters.end(), 0);
  thrust::device_vector<float> mins(N); // minimum discance for each vector
  thrust::device_vector<float> vec_modulus_k(N * k); // vector filled with k to get the number of cluster from long array
  thrust::device_vector<int> indices(N); // id-s of vectors from sorted clusters
  thrust::device_vector<int> clusterSizes(k); // number of vectors in each cluster 
  thrust::device_vector<float> vectorsInCluster(n); // vectors in actual cluster (will be resized, but we always have >= 1)
  thrust::device_vector<float> actual_indices(1); // range from indices for actual cluster 
  thrust::device_vector<int> data_starts(k); // starts of segments from indices
  thrust::device_vector<int> data_ends(k); // ends of segments from indices
  thrust::device_vector<bool> docopy(N * n); // binary mask
  thrust::device_vector<float> fcol_sums(n);
  
  // CPU - GPU copying
  thrust::device_vector<float> d_data(data, data + n * N);
  thrust::device_vector<float> d_centr(cs, cs + n * k);

  while (numIters < MAX_ITERATIONS && (float)delta / (float)N > EPS) {
    delta = 0;
    
    // distance calculation
    calculateDistances(n, N, k, d_data, d_centr, tmp_distances);
   hipDeviceSynchronize();
    // nearest centroid searching
    findNearestCentroid(k, N, d_centr, tmp_distances, mins, vec_modulus_k, d_clusters);
    
    // cluster changes counting
    countClusterChanges(delta, old_d_clusters, d_clusters);

    thrust::copy(d_clusters.begin(), d_clusters.end(),
                 old_d_clusters.begin()); // preprocessing

    
    // new centorids computation
    findNewCentroids(n, N, k, d_data, d_centr, indices, d_clusters,
                     clusterSizes, data_starts, data_ends, vectorsInCluster,
                     actual_indices, fcol_sums, docopy);
    ++numIters;
  }
  clstrs = new int[old_d_clusters.size()];
  thrust::copy(old_d_clusters.begin(), old_d_clusters.end(), clstrs);
}

void writeDataToFile(float* data, const int* clusters, int N, int n) {
    std::ofstream outputFile;
    outputFile.open("out_thrust.txt");
    if (outputFile.is_open()) {
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < n; ++j) {
                outputFile << data[i*n + j] << ' ';
            }
            outputFile << clusters[i] << '\n';
        }
        outputFile.close();
        std::cout << "Data written successfully \n" << std::endl;
    } else {
        std::cout << "Unable to open the file \n"<< std::endl;
    }
}

void readFile(std::istream &inputFile, int &N, int &n, int &k, float *&data,
              float *&centroids) {
  std::string inputString;
  getline(inputFile, inputString);
  N = atoi(inputString.c_str()); 
  getline(inputFile, inputString);
  n = atoi(inputString.c_str()); 
  getline(inputFile, inputString);
  k = atoi(inputString.c_str()); 

  data = new float[N * n];
  centroids = new float[k * n];
  float value = 0.0f;
  int ind = 0;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    value = 0.0f;
    while (iss >> value) {
      data[ind] = value;
      if (ind < k * n) {
        centroids[ind] = value;
      }
      ++ind;
    }
  }
}

int main(int argc, char **argv) {
  // file validation
  std::string inFile = "";
  if (argc == 2) {
    inFile = argv[1];
  } else {
    std::cout << "Usage: ./cufile InputFile \n";
    return 1;
  }
  std::ifstream inputFile;
  inputFile.open(inFile.c_str(), std::ios::in);
  if (!inputFile.is_open()) {
    std::cout << "Error opening file: " << inFile << std::endl;
    return 1;
  }

  // data declaration
  float *data; // vectors
  float *centroids; // new centroids
  int *clusters; // clusters[i] == old cluster number for i-th vector
  int N, n, k;
  readFile(inputFile, N, n, k, data, centroids);
  inputFile.close();

  // K-means clusterization
  KMeansClustering(data, centroids, clusters, k, n, N, 1);

  writeDataToFile(data, clusters, N, n);

  delete[] data;
  delete[] centroids;
  delete[] clusters;
  return 0;
}