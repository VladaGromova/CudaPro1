#include <cfloat>
#include <climits>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#pragma hd_warning_disable
//#define FILENAME "data.txt"
//#define FILENAME "points_generated.txt"
#define FILENAME "myData.txt"
//#define FILENAME "cluster_data.txt"

#define BLOCK_SIZE 16
#define MAX_ITERATIONS 100
#define EPS 0.000001f
//#define MAX_THREADS_IN_BLOCK 512

#define MAX_THREADS_IN_BLOCK 16

int main() {
  std::ifstream inputFile(FILENAME);
  std::string inputString;
  getline(inputFile, inputString);
  int N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  int n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  int k = atoi(inputString.c_str()); // real B width, real C width

  std::vector<thrust::device_vector<float>> pointsArray(n); // Each vector represents a dimension
  std::vector<thrust::device_vector<float>> centroidsArray(n); // Each vector represents a dimension

  float value = 0.0f;
  int index = 0;
  int indexOfVector = 0;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    value = 0.0f;
    index = 0;
    while (iss >> value) {
      pointsArray[index].push_back(value);
      if(indexOfVector < k){
        centroidsArray[index].push_back(value);
      }
      ++index;
    }
    ++indexOfVector;
  }

    std::cout << "Points Array:" << std::endl;
    for (int i = 0; i < n; ++i) {
        std::cout << "Dimension " << i << ": ";
        for (float val : pointsArray[i]) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

    // Print the contents of centroidArray
    std::cout << "\nCentroids Array:" << std::endl;
    for (int i = 0; i < n; ++i) {
        std::cout << "Dimension " << i << ": ";
        for (float val : centroidsArray[i]) {
            std::cout << val << " ";
        }
        std::cout << std::endl;
    }

  inputFile.close();

  std::cout<<"\nBye!\n";
  return 0;
}