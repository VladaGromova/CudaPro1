#include "hip/hip_runtime.h"
#include <cfloat>
#include <climits>
//#include <concurrencysal.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iterator>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include <cstdio>
#include <math.h>

#include <time.h>
#include <sys/time.h>
#include <stdlib.h>

#pragma hd_warning_disable
#define FILENAME "data.txt"
//#define FILENAME "points_generated.txt"
//#define FILENAME "myData.txt"
//#define FILENAME "cluster_data.txt"

#define EPS 0.000001f

unsigned long long dtime_usec(unsigned long long prev){
#define USECPSEC 1000000ULL
  timeval tv1;
  gettimeofday(&tv1,0);
  return ((tv1.tv_sec * USECPSEC)+tv1.tv_usec) - prev;
}

struct dkeygen : public thrust::unary_function<int, int>
{
  int dim;
  int numd;

  dkeygen(const int _dim, const int _numd) : dim(_dim), numd(_numd) {};

  __host__ __device__ int operator()(const int val) const {
    return (val/dim);
    }
};


struct clusterkeygen : public thrust::unary_function<int, int>
{
  int stride;

  clusterkeygen(const int _stride) : stride(_stride) {};

  __host__ __device__ int operator()(const int val) const {
    return (val % stride);
    }
};


typedef thrust::tuple<float, float> mytuple;
struct my_dist : public thrust::unary_function<mytuple, float>
{
  __host__ __device__ float operator()(const mytuple &my_tuple) const {
    float temp = thrust::get<0>(my_tuple) - thrust::get<1>(my_tuple);
    return temp*temp;
  }
};


struct MinWithIndex {
    __host__ __device__
    thrust::tuple<float, int> operator()(const thrust::tuple<float, int>& a, const thrust::tuple<float, int>& b) const {
        return (thrust::get<0>(a) < thrust::get<0>(b)) ? a : b;
    }
};


struct d_idx : public thrust::unary_function<int, int>
{
  int dim;
  int numd;

  d_idx(int _dim, int _numd) : dim(_dim), numd(_numd) {};

  __host__ __device__ int operator()(const int val) const {
    return (val % (dim*numd));
    }
};



struct c_idx : public thrust::unary_function<int, int>
{
  int dim;
  int numd;

  c_idx(int _dim, int _numd) : dim(_dim), numd(_numd) {};

  __host__ __device__ int operator()(const int val) const {
    return (val % dim) + (dim * (val/(dim*numd)));
    }
};

struct my_sqrt : public thrust::unary_function<float, float>
{
  __host__ __device__ float operator()(const float val) const {
    return sqrtf(val);
  }
};

template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T>
{
  T C; // number of columns
  
  __host__ __device__
  linear_index_to_row_index(T C) : C(C) {}

  __host__ __device__
  T operator()(T i)
  {
    return i / C;
  }
};

struct sum_functor {
    __host__ __device__
    float operator()(const float &a, const float &b) const {
        return a + b;
    }
};

struct div_functor : public thrust::unary_function<int,int>
{
    int m;
    div_functor(int _m) : m(_m) {};

    __host__ __device__
    int operator()(int x) const
    {
        return x / m;
    }
};

struct is_true
{
    __host__ __device__
    bool operator()(bool x) { return x; }
};

unsigned long long eucl_dist_thrust(thrust::host_vector<float> &cs, thrust::host_vector<float> &data, thrust::host_vector<float> &dist, int k, int n, int N, int print){

  thrust::device_vector<float> d_data = data;
  thrust::device_vector<float> d_centr = cs;
  thrust::device_vector<float> values_out(k*N);

  unsigned long long compute_time = dtime_usec(0);

thrust::reduce_by_key(
    // keys: 0...0 1...1 ... k*n*N
    thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0), dkeygen(n, N)), // begining of input key range
    thrust::make_transform_iterator(thrust::make_counting_iterator<int>(n*N*k), dkeygen(n, N)), // end of input key range
    thrust::make_transform_iterator(thrust::make_zip_iterator( // begining of values range - tu chcemy miec odleglosci
      thrust::make_tuple(
        thrust::make_permutation_iterator(
          d_centr.begin(), 
          thrust::make_transform_iterator(
              thrust::make_counting_iterator<int>(0), d_idx(n, k)
          )
        ),
        thrust::make_permutation_iterator(
          d_data.begin(), 
          thrust::make_transform_iterator(
            thrust::make_counting_iterator<int>(0), c_idx(n, k)
          )
        )
      )
     ), my_dist()),
    thrust::make_discard_iterator(), // keys output (nie potrzebujemy tego)
    values_out.begin()    // values output - wynik
    );

  thrust::transform(values_out.begin(), values_out.end(), values_out.begin(), my_sqrt());
  hipDeviceSynchronize();
 compute_time = dtime_usec(compute_time);
std:: cout<<"Distances :\n";
  if (print){
    thrust::copy(values_out.begin(), values_out.end(), std::ostream_iterator<float>(std::cout, ", "));
    std::cout << std::endl;
    }
  thrust::copy(values_out.begin(), values_out.end(), dist.begin());

int numColumns = k; // Number of columns
    thrust::device_vector<float> mins(N);
    thrust::device_vector<int> d_clusters(N);

    // Perform reduction to find minimum value and its position for each row
    thrust::reduce_by_key(
        thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(numColumns)),
        thrust::make_transform_iterator(thrust::counting_iterator<int>(k*N), linear_index_to_row_index<int>(numColumns)),
        thrust::make_zip_iterator(thrust::make_tuple(values_out.begin(), thrust::counting_iterator<int>(0))),
        thrust::make_discard_iterator(), // Discard keys output
        thrust::make_zip_iterator(thrust::make_tuple(mins.begin(), d_clusters.begin())),
        thrust::equal_to<int>(),
        MinWithIndex()
    );
    
thrust::device_vector<float> V2(N*k);
thrust::fill(V2.begin(), V2.end(), k);
thrust::transform(d_clusters.begin(), d_clusters.end(), V2.begin(), d_clusters.begin(), thrust::modulus<int>());

 std:: cout<<"\nClusters:\n";
   thrust::copy_n(d_clusters.begin(),d_clusters.end(),std::ostream_iterator<int>(std::cout, ", "));
   std::cout << std::endl;


 std:: cout<<"\nMins:\n";
   thrust::copy_n(mins.begin(),mins.end(),std::ostream_iterator<float>(std::cout, ", "));
   std::cout << std::endl;


  thrust::device_vector<int> indices(N);
    thrust::sequence(indices.begin(), indices.end());
    thrust::sort_by_key(d_clusters.begin(), d_clusters.end(), indices.begin());

 std:: cout<<"\nIndices:\n";
   thrust::copy_n(indices.begin(),indices.end(),std::ostream_iterator<float>(std::cout, ", "));
   std::cout << std::endl;

    // Oblicz liczbę wystąpień każdego klastra
    thrust::device_vector<int> clusterSizes(k);
    thrust::reduce_by_key(
        d_clusters.begin(), d_clusters.end(),
        thrust::make_constant_iterator(1),
        thrust::make_discard_iterator(),
        clusterSizes.begin(),
        thrust::equal_to<int>(),
        thrust::plus<int>()
    );
     std:: cout<<"\nCluster sizes:\n";
   thrust::copy_n(clusterSizes.begin(),clusterSizes.end(),std::ostream_iterator<float>(std::cout, ", "));
   std::cout << std::endl;


thrust::fill(d_centr.begin(), d_centr.end(), 0.0);
thrust::device_vector<float> vectorsInCluster(n);
thrust:: device_vector<float> actual_indices(1);
thrust::device_vector<int> data_starts(k);
thrust::device_vector<int> data_ends(k);
thrust::exclusive_scan(clusterSizes.begin(), clusterSizes.end(), data_starts.begin()); 
thrust::inclusive_scan(clusterSizes.begin(), clusterSizes.end(), data_ends.begin()); 
std:: cout<<"\n Data starts:\n";
thrust::copy_n(data_starts.begin(),data_starts.end(),std::ostream_iterator<int>(std::cout, ", "));
std::cout << std::endl;
std:: cout<<"\n Data ends:\n";
thrust::copy_n(data_ends.begin(),data_ends.end(),std::ostream_iterator<int>(std::cout, ", "));
std::cout << std::endl;
thrust::device_vector<bool> docopy(N*n);

int i=0;
//for(int i=0; i<k; ++i){
  vectorsInCluster.resize(clusterSizes[i] * n);
  actual_indices.resize(clusterSizes[i]);
  thrust::copy(indices.begin() + data_starts[i], indices.end() + data_ends[i], actual_indices.begin());

  typedef thrust::counting_iterator<int> counter;
    typedef thrust::transform_iterator<div_functor, counter> rowIterator;
    rowIterator rows_begin = thrust::make_transform_iterator(thrust::make_counting_iterator(0), div_functor(n));
    rowIterator rows_end = rows_begin + (N*n);
  thrust::binary_search(actual_indices.begin(), actual_indices.end(), rows_begin, rows_end, docopy.begin());
  thrust::copy_if(thrust::make_counting_iterator<int>(0), 
                  thrust::make_counting_iterator<int>(N*n),
                  docopy.begin(), 
                  vectorsInCluster.begin(), 
                  is_true()
  );
  std:: cout<<"\n Actrual vectors:\n";
thrust::copy_n(vectorsInCluster.begin(),vectorsInCluster.end(),std::ostream_iterator<float>(std::cout, ", "));
std::cout << std::endl;
//}


    // // Podziel sumę przez liczbę wystąpień, aby otrzymać centroidy
    // thrust::transform(
    //     centroids.begin(), centroids.end(),
    //     clusterSizes.begin(),
    //     centroids.begin(),
    //     thrust::divides<float>()
    // );


  return compute_time;
}

int main() {
  std::ifstream inputFile(FILENAME);
  std::string inputString;
  getline(inputFile, inputString);
  long N = atoi(inputString.c_str()); // real A height, real C height
  getline(inputFile, inputString);
  int n = atoi(inputString.c_str()); // real A width, real B height
  getline(inputFile, inputString);
  int k = atoi(inputString.c_str()); // real B width, real C width

  float* data = new float[N*n];
  float* centroids = new float[k*n];

  float value = 0.0f;
  int ind = 0;
  while (getline(inputFile, inputString)) {
    std::istringstream iss(inputString);
    value = 0.0f;
    while (iss >> value) {
      data[ind] = value;
      if(ind < k*n){
        centroids[ind]  = value;
      }
      ++ind;
    }
  }
  inputFile.close();
  std::cout<<"Data: \n";
  for (int i = 0; i<N; ++i) {
    for(int j=0; j< n; ++j){
      std::cout<< data[i*n + j]<< ' ';
    }
    std::cout<<'\n';
  }
    std::cout<<"\nCentroids: \n";
  for (int i = 0; i<k; ++i) {
    for(int j=0; j< n; ++j){
      std::cout<< data[i*n + j]<< ' ';
    }
    std::cout<<'\n';
  }
  
  thrust::host_vector<float> h_data(data, data + N*n);
  thrust::host_vector<float> h_centr(centroids, centroids + k*n);
  thrust::host_vector<float> h_dist(k*N);
  eucl_dist_thrust(h_centr, h_data, h_dist, k, n, N, 1);
    
  std::cout<<"\nBye!\n";
  return 0;
}